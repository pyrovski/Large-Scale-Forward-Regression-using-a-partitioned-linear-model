#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include <hipblas.h>
#include <sstream>
#include <iostream>

#include "type.h"

#include "plm.h"

#ifdef _DEBUG
#if __CUDA_ARCH__ >= 200
//#define printGPU
#endif
#endif

__device__ int printBIDs(unsigned BID){
  return(BID == 0 || BID == 40);
}

#include "cuda_blas.cu"

using namespace std;
//__shared__ double fval; // scalar
extern __shared__ double shared[];

__constant__ double d_Xty[iterationLimit + 27];

//const unsigned GPitch = (iterationLimit + 27) % 16 ? iterationLimit + 27 : (iterationLimit + 27)/16 * 16 + 16;

// extra space for padding.  Could make this triangular.
__constant__ double d_G[(iterationLimit + 27)*(iterationLimit + 27)];

__global__ void plm(// inputs
		    const unsigned m,          // rows of X
		    //const unsigned n,        // colums of X == number of blocks
		    const double *snptsnp,      // scalar, unique to block
		    const double *Xtsnp,        // n x 1 vector, unique to block
		    const unsigned XtsnpPitch, 
		    const double errorSS,       // scalar
		    const unsigned errorDF,       // scalar
		    //const double *G,          // symmetric matrix in const mem
		    //const double *Xty,        // n x 1 vector in const mem
		    const double *snpty,        // scalar, unique to block
		    const unsigned *snpMask,   // n x 1 vector
		    // outputs
		    double *f){
  /*! @todo could compute two SNPs per thread block.  
    This would ease the limitation of 8 thread blocks/MP for SM 1.3 devices,
    but might need some thread padding for warps.
   */

  double *reduce = shared; // n x 1
  //double *reduce2 = reduce + n;
  double GtXtsnp; // each thread stores one element of each array // Xtsnp
  //! @todo these might use fewer registers if kept in shared memory
  double snptmy; // scalar
  double s; // scalar

  unsigned BID = blockIdx.x + gridDim.x * blockIdx.y;
  unsigned TID = threadIdx.x;
  if(snpMask[BID]){
    // don't compute new F
    if(!TID)
      f[BID] = 0;
    return; 
  }
  // snptsnp - snptXGXtsnp:


  // GtXtsnp
  GtXtsnp = vecGMatCSq(TID, Xtsnp + BID * XtsnpPitch/sizeof(double), blockDim.x, d_G, 
		     blockDim.x,  //! length of column plus padding (no padding)
		     reduce); 
  
  // snptsnp - snptXGXtsnp
  dotRG(TID, blockDim.x, GtXtsnp, Xtsnp + BID * XtsnpPitch/sizeof(double), reduce);
  s = snptsnp[BID] - *reduce;
#ifdef printGPU
  if(printBIDs(BID)){
    printf("b%03u\tt%03u\tXtsnp: %1.10le\n", BID, TID, Xtsnp[BID * XtsnpPitch/sizeof(double) + TID]);
    printf("b%03u\tt%03u\tGtXtsnp: %1.10le\n", BID, TID, GtXtsnp);
    if(!TID){
      printf("b%03u\tt%03u\tsnptsnp: %1.10le\n", BID, TID, snptsnp[BID]);
      printf("b%03u\tt%03u\tsnptXGXtsnp: %1.10le\n", BID, TID, *reduce);
      printf("b%03u\tt%03u\ts: %1.10le\n", BID, TID, s);
    }
  }
#endif
  // 1/(above)
  if(s > doubleTol){
    s = (double)1/s;
    
    // snptmy
    dotRG(TID, blockDim.x, GtXtsnp, d_Xty, reduce);
    snptmy = -*reduce;
    
    if(!TID){
#ifdef printGPU
      if(printBIDs(BID)){
	printf("b%03u\tt%03u\tsnptXGXty: %1.10le\n", BID, TID, snptmy);
	printf("b%03u\tt%03u\tsnpty: %1.10le\n", BID, TID, snpty[BID]);
      }
#endif
      snptmy += snpty[BID];
      double modelSS = snptmy * snptmy * s;
      double errorSS2 = errorSS - modelSS;
      unsigned V2 = errorDF - 1;
      f[BID] = modelSS / errorSS2 * V2;
#ifdef printGPU
  if(printBIDs(BID)){

    printf("b%03u\tt%03u\tmodelSS: %1.10le\n", BID, TID, modelSS);
    printf("b%03u\tt%03u\tnew errorSS: %1.10le\n", BID, TID, errorSS2);
    printf("b%03u\tt%03u\tnew V2: %u\n", BID, TID, V2);
    printf("b%03u\tt%03u\tf: %1.10le\n", BID, TID, f[BID]);
  }
#endif

    }
    return;
  } else {
    if(!TID)
      f[BID] = 0;
    return;
  }
}

hipEvent_t start, stopKernel, stopMax;

unsigned plm_GPU(unsigned geno_count, unsigned blockSize, 
		 unsigned m, double* d_snptsnp, double* d_Xtsnp, 
		 unsigned d_XtsnpPitch, double ErrorSS, unsigned V2, 
		 double* d_snpty, unsigned* d_snpMask, double* d_f) throw(int)
{
    cublasGetError();
    hipEventRecord(start, 0);
    plm<<<geno_count, blockSize, blockSize * sizeof(double)>>>
      (m ,        
       d_snptsnp, 
       d_Xtsnp, 
       d_XtsnpPitch, 
       ErrorSS, V2, 
       d_snpty, 
       d_snpMask,
       d_f);
    hipEventRecord(stopKernel, 0);
    cutilSafeCall(hipDeviceSynchronize());
    // cublas uses 1-based index
    unsigned maxFIndex = hipblasIdamax(geno_count, d_f, 1);
    hipEventRecord(stopMax, 0);
    if(!maxFIndex){
      cerr << "maxFIndex <= 0!" << endl;
      throw(1);
    }
    maxFIndex -= 1;

    return maxFIndex;
}

/*!
  should only be called once
 */
void copyToDevice(unsigned geno_count, const unsigned n, 
		 double *&d_snptsnp, double *&d_Xtsnp, size_t &d_XtsnpPitch, 
		 double *&d_snpty, unsigned *&d_snpMask, double *&d_f,
		 const vector<double> &SNPtSNP, const FortranMatrix &XtSNP,
		 const vector<double> &SNPty,
		 const vector<double> &Xty, const FortranMatrix &XtXi, 
		 const vector<unsigned> &snpMask){
  hipEventCreate(&start);
  hipEventCreate(&stopKernel);
  hipEventCreate(&stopMax);

  cutilSafeCall(hipMalloc(&d_snpMask, geno_count * sizeof(unsigned)));
  cutilSafeCall(hipMemcpy(d_snpMask, &snpMask[0], 
			   geno_count * sizeof(unsigned), 
			   hipMemcpyHostToDevice));

  
  //! @todo this won't be coalesced
  cutilSafeCall(hipMalloc(&d_snptsnp, geno_count * sizeof(double)));
  cutilSafeCall(hipMemcpy(d_snptsnp, &SNPtSNP[0], geno_count * sizeof(double), 
			   hipMemcpyHostToDevice));

  cutilSafeCall(hipMallocPitch(&d_Xtsnp, &d_XtsnpPitch, 
				(n + iterationLimit) * sizeof(double), 
				geno_count));
  cutilSafeCall(hipMemcpy2D(d_Xtsnp, d_XtsnpPitch, &XtSNP.values[0], 
			     n * sizeof(double), n * sizeof(double), geno_count, 
			     hipMemcpyHostToDevice));
  
  cutilSafeCall(hipMalloc(&d_snpty, geno_count * sizeof(double)));
  cutilSafeCall(hipMemcpy(d_snpty, &SNPty[0], geno_count * sizeof(double), 
			   hipMemcpyHostToDevice));
  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_G), &XtXi.values[0], n * n * sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Xty), &Xty[0], n * sizeof(double)));
  
  cutilSafeCall(hipMalloc(&d_f, geno_count * sizeof(double)));

}

void copyUpdateToDevice(unsigned id, unsigned iteration,  
			unsigned geno_count, unsigned n,
		       unsigned *d_snpMask, 
		       int maxFIndex, double *d_Xtsnp, 
		       size_t d_XtsnpPitch,
		       const vector<unsigned> &snpMask,
		       FortranMatrix &XtSNP, const FortranMatrix &XtXi,
		       const vector<double> &Xty){
  if(maxFIndex >= 0){
#ifdef _DEBUG
      cout << "iteration " << iteration << " id " << id 
	   << " masking index " << maxFIndex << " on device: " 
	   << snpMask[maxFIndex] << endl;
#endif
    cutilSafeCall(hipMemcpy(d_snpMask + maxFIndex, &snpMask[maxFIndex], 
			     sizeof(unsigned), hipMemcpyHostToDevice));
    unsigned maskVal;
    cutilSafeCall(hipMemcpy(&maskVal, d_snpMask + maxFIndex,
			     sizeof(unsigned), hipMemcpyDeviceToHost));
#ifdef _DEBUG
      cout << "iteration " << iteration << " id " << id 
	   << " mask index " << maxFIndex << ": "
	   << maskVal << endl;
#endif
      /* this doesn't help...
    cutilSafeCall(hipMemcpy(d_snpMask, &snpMask[0], 
			     sizeof(unsigned) * geno_count, hipMemcpyHostToDevice));
      */
  }

    //! copy updated XtSNP to GPU
    cutilSafeCall(hipMemcpy2D(d_Xtsnp + n, 
			       d_XtsnpPitch,
			       &XtSNP(n, (unsigned)0), 
			       (n + 1) * sizeof(double),
			       sizeof(double),
			       geno_count,
			       hipMemcpyHostToDevice));

    // update GPU G (const mem)
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_G), &XtXi.values[0], 
				     (n + 1) * (n + 1) * sizeof(double)));

    // update GPU Xty (const mem)
    // call fails unless we update the whole thing
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Xty), &Xty[0], (n + 1) * sizeof(double)));

}

float getGPUCompTime(){
  float computation_elapsed_time;
  hipEventElapsedTime(&computation_elapsed_time, start, stopKernel);
  return computation_elapsed_time / 1000.0f;
}

float getGPUMaxTime(){
  float computation_elapsed_time;
  hipEventElapsedTime(&computation_elapsed_time, stopKernel, stopMax);
  return computation_elapsed_time / 1000.0f;
}

void getMaxF(unsigned id, unsigned iteration, unsigned geno_count, 
	     vector<double> &Fval, 
	     unsigned maxFIndex, double *d_f){
#ifndef _DEBUG
    cutilSafeCall(hipMemcpy(&Fval[maxFIndex], &d_f[maxFIndex], sizeof(double),
			     hipMemcpyDeviceToHost));
#else
    cutilSafeCall(hipMemcpy(&Fval[0], d_f, geno_count * sizeof(double),
			     hipMemcpyDeviceToHost));
    {
      stringstream ss;
      ss << "Fval_" << iteration << "_" << id << ".dat";
      writeD(ss.str(), Fval);
    }

#endif
}
