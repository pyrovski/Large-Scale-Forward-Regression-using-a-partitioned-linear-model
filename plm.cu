#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include <hipblas.h>
#include <sstream>
#include <iostream>

#include "type.h"
#include "fortran_matrix.h"
#include "reference_glm.h"

#include "plm.h"

#ifdef _DEBUG
#if __CUDA_ARCH__ >= 200
#define printGPU
#endif
#endif

__device__ int printBIDs(unsigned BID){
  return(BID == 0);
}

void initGrid(dim3 &grid, unsigned geno_count) throw(int){
  static unsigned old_geno_count = 0;
  static dim3 oldGrid;
  if(old_geno_count == geno_count){
    grid = oldGrid;
    return;
  }
  grid.x = geno_count;
  grid.y = 1;
  grid.z = 1;
  while(grid.x > 65535){
    grid.x += grid.x % 2;
    grid.x /= 2;
    grid.y *= 2;
  }
  // could probably do a better job factoring here; instead bail
  if(grid.y > 65535)
    throw(2);
  oldGrid = grid;
#ifdef _DEBUG
  std::cout << "grid: " << grid.x << "x" << grid.y << std::endl;
#endif
}

#include "cuda_blas.cu"

using namespace std;
//__shared__ double fval; // scalar
extern __shared__ double shared[];

__constant__ double d_Xty[fixedPlusIteration_limit + 1];

/*! extra space for padding.  Could make this triangular.
  If triangular, we could fit fixedPlusIteration_limit = 127, rather than 89
 */
__constant__ double d_G[(fixedPlusIteration_limit + 1)*(fixedPlusIteration_limit + 1)];
//! @todo should y be in constant mem?  Not if we use cublas for dgemv (SNPty)

__global__ void plm(// inputs
		    const unsigned geno_count, // # of SNPs == # of blocks?
		    const unsigned m,          // rows of X
		    //const unsigned n,        // colums of X == threads/block
		    const double *snptsnp,      // scalar, unique to block
		    const double *Xtsnp,        // n x 1 vector, unique to block
		    const unsigned XtsnpPitchInWords, 
		    const double errorSS,       // scalar
		    const unsigned errorDF,       // scalar
		    //const double *G,          // symmetric matrix in const mem
		    //const double *Xty,        // n x 1 vector in const mem
		    const double *snpty,        // scalar, unique to block
		    const char *snpMask,   // n x 1 vector
		    // outputs
		    float *f){
  /*! @todo could compute two SNPs per thread block.  
    This would ease the limitation of 8 thread blocks/MP for SM 1.3 devices,
    but might need some thread padding for warps.
   */

  double *reduce = shared; // n x 1
  double GtXtsnp; // each thread stores one element of each array // Xtsnp
  //! @todo these might use fewer registers if kept in shared memory
  double snptmy; // scalar

  double s; // scalar

  unsigned BID = blockIdx.x + gridDim.x * blockIdx.y;
  unsigned TID = threadIdx.x;

  #ifdef printGPU
  if(!TID && printBIDs(BID))
     printf("BID: %u\n", BID);
  #endif

  if(BID >= geno_count)
    return;
  if(snpMask[BID]){
    // don't compute new F
    if(!TID)
      f[BID] = 0;
    return; 
  }
  // snptsnp - snptXGXtsnp:

  //#error fixme double read
  double myXtsnp = *(Xtsnp + BID * XtsnpPitchInWords + TID);
  // GtXtsnp
  GtXtsnp = vecRMatCSq(TID, myXtsnp, blockDim.x, d_G, 
		       blockDim.x);  //! length of column plus padding (no padding) 
  
  // snptsnp - snptXGXtsnp
  dotRR(TID, blockDim.x, GtXtsnp, myXtsnp, reduce);
  s = snptsnp[BID] - *reduce;
#ifdef printGPU
  if(printBIDs(BID)){
    for(int i = 0; i < blockDim.x; i++){
      if(i == TID){
	printf("b%03u\tt%03u\tXtsnp: %1.10le\n", BID, TID, Xtsnp[BID * XtsnpPitchInWords + TID]);
	for(int j = 0; j < blockDim.x; j++)
	  printf("b%03u\tt%03u\tG[%d,%d]: %1.10le\n", BID, TID, i, j, d_G[i*blockDim.x + j]);
	printf("b%03u\tt%03u\tGtXtsnp: %1.10le\n", BID, TID, GtXtsnp);
	if(!TID){
	  printf("b%03u\tt%03u\tsnptsnp: %1.10le\n", BID, TID, snptsnp[BID]);
	  printf("b%03u\tt%03u\tsnptXGXtsnp: %1.10le\n", BID, TID, shared[0]);
	  printf("b%03u\tt%03u\ts: %1.10le\n", BID, TID, s);
	}
      }
      __syncthreads();
    }
  }
#endif
  // 1/(above)
  if(s > doubleTol){
    s = (double)1/s;
    
    // snptmy
    dotRG(TID, blockDim.x, GtXtsnp, d_Xty, reduce);
    snptmy = -*reduce;
    
    if(!TID){
#ifdef printGPU
      if(printBIDs(BID)){
	for(int i = 0; i < blockDim.x; i++){
	  if(i == TID){
	    printf("b%03u\tt%03u\tsnptXGXty: %1.10le\n", BID, TID, snptmy);
	    printf("b%03u\tt%03u\tsnpty: %1.10le\n", BID, TID, snpty[BID]);
	  }
	  __syncthreads();
	}
      }
#endif
      snptmy += snpty[BID];
      //! @todo this could be single precision
      float modelSS = snptmy * snptmy * s;

      double errorSS2 = errorSS - (double)modelSS;

      unsigned V2 = errorDF - 1;
      f[BID] = modelSS / errorSS2 * V2;
#ifdef printGPU
  if(printBIDs(BID)){
    for(int i = 0; i < blockDim.x; i++){
      if(i == TID){
	
	printf("b%03u\tt%03u\tmodelSS: %1.10le\n", BID, TID, modelSS);
	printf("b%03u\tt%03u\tnew errorSS: %1.10le\n", BID, TID, errorSS2);
	printf("b%03u\tt%03u\tnew V2: %u\n", BID, TID, V2);
	printf("b%03u\tt%03u\tf: %1.10le\n", BID, TID, f[BID]);
      }
      __syncthreads();
    }
  }
#endif

    }
    return;
  } else {
    if(!TID)
      f[BID] = 0;
    return;
  }
}

hipEvent_t start, stopKernel, stopMax;

unsigned plm_GPU(unsigned geno_count, unsigned blockSize, 
		 unsigned m, double* d_snptsnp, double* d_Xtsnp, 
		 unsigned d_XtsnpPitch, double ErrorSS, unsigned V2, 
		 double* d_snpty, char* d_snpMask, float* d_f,
		 vector<float> &Fval) throw(int)
{
    cublasGetError();
    hipEventRecord(start, 0);
    dim3 grid;
    initGrid(grid, geno_count);
    
    plm<<<grid, blockSize, blockSize * sizeof(double)>>>
      (geno_count,
       m,
       d_snptsnp, 
       d_Xtsnp, 
       d_XtsnpPitch / sizeof(double), 
       ErrorSS, V2, 
       d_snpty, 
       d_snpMask,
       d_f);
    hipEventRecord(stopKernel, 0);

#ifdef _DEBUG
    cutilSafeCall(hipDeviceSynchronize());
    cutilSafeCall(hipMemcpy(&Fval[0], d_f, geno_count * sizeof(float),
			     hipMemcpyDeviceToHost));
#endif

    hipblasStatus_t status = cublasGetError();

    // cublas uses 1-based index
    int maxFIndex = hipblasIsamax(geno_count, d_f, 1);
    hipEventRecord(stopMax, 0);
    status = cublasGetError();
    if(status != HIPBLAS_STATUS_SUCCESS){
      cerr << "cublas error in hipblasIdamax(): " << status << endl;
      throw(1);
    }
    if(maxFIndex <= 0){
      cerr << "maxFIndex <= 0:" << maxFIndex << endl;
      throw(1);
    }
    maxFIndex -= 1;

    return maxFIndex;
}

//! @todo convert for gpu
/*!
  should only be called once
 */
int copyToDevice(const unsigned id, // MPI rank
		 const unsigned verbosity,
		 const unsigned geno_count, // # of SNPs
		 const unsigned n, // # of columns in X
		 const unsigned geno_ind, // # length of each SNP
		 double *&d_snptsnp, double *&d_Xtsnp, size_t &d_XtsnpPitch, 
		 double *&d_snpty, char *&d_snpMask, float *&d_f, 
		 double *&d_geno, size_t &d_genoPitch, 
		 double *&d_Xt, size_t &d_XtPitch,
		 double *&d_y,
		 const vector<double> &Xty, 
		 const FortranMatrix &XtXi, // G
		 const FortranMatrix &Xt, // only for XtSNP
		 const FortranMatrix &geno,
		 double *&d_nextSNP,
		 const double *y){

  uint64_t snpMaskSize = geno_count * sizeof(char), 
    snptsnpSize = geno_count * sizeof(double), 
    XtsnpSize, 
    snptySize = geno_count * sizeof(double), 
    fSize = geno_count * sizeof(float),
    genoSize, // new
    XtSize, // new
    nextSNPSize = geno_ind * sizeof(double); // new

  uint64_t totalSize;

  
  cutilSafeCall(hipMallocPitch(&d_Xtsnp, &d_XtsnpPitch, 
				(n + iterationLimit) * sizeof(double), 
				geno_count));
  XtsnpSize = d_XtsnpPitch * geno_count;
   
  cutilSafeCall(hipMallocPitch(&d_geno, &d_genoPitch, geno_ind * sizeof(double), geno_count));
  if(d_genoPitch < geno_ind){
    cerr << "error allocating d_geno" << endl;
    return -1;
  }
  genoSize = geno_count * d_genoPitch;

  //! store Xt
  cutilSafeCall(hipMallocPitch(&d_Xt, &d_XtPitch, n, geno_ind));
  if(d_XtPitch < n){
    cerr << "error allocating d_Xt" << endl;
    return -1;
  }
  XtSize = geno_ind * d_XtPitch;

  // Xty and G are in constant memory; compiler checks this
  totalSize = fSize + XtsnpSize + snptsnpSize + snpMaskSize + snptySize
    + genoSize + XtSize + nextSNPSize;

  struct hipDeviceProp_t prop;
  hipError_t cudaStatus;
  int device;
  hipGetDevice(&device);
  cudaStatus = hipGetDeviceProperties(&prop, device);
  if(cudaStatus != hipSuccess){
    cerr << "id " << id << " error in hipGetDeviceProperties()" << endl;
    return -1;
  }
  if(verbosity > 1)
    cout << "id " << id << " requires " << totalSize << "/" 
	 << prop.totalGlobalMem << " bytes global memory"  
	 << " (" << (100.0 * totalSize) / prop.totalGlobalMem 
	 << "%)"
	 << endl;
  if(totalSize >= prop.totalGlobalMem){
    cerr << "id " << id << " insufficient device memory" << endl;
    return -1;
  }

  cutilSafeCall(hipMemcpy2D(d_geno, d_genoPitch, &geno.values[0], 
			     geno_ind * sizeof(double), 
			     geno_ind * sizeof(double),
			     geno_count, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy2D(d_Xt, d_XtPitch, &Xt.values[0],
			     n * sizeof(double), 
			     n * sizeof(double),
			     geno_ind, hipMemcpyHostToDevice));
  cutilSafeCall(hipMalloc(&d_snpMask, geno_count * sizeof(char)));

  cutilSafeCall(hipMemsetAsync(d_snpMask, 0, geno_count * sizeof(char)));
  
  //! @todo these won't be coalesced; each block reads one value
  cutilSafeCall(hipMalloc(&d_snptsnp, geno_count * sizeof(double)));  
  cutilSafeCall(hipMalloc(&d_snpty, geno_count * sizeof(double)));
  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_G), &XtXi.values[0], 
				   n * n * sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Xty), &Xty[0], n * sizeof(double)));
  
  cutilSafeCall(hipMalloc(&d_f, geno_count * sizeof(float)));
  cutilSafeCall(hipMalloc(&d_nextSNP, geno_ind * sizeof(double)));

  cutilSafeCall(hipMalloc(&d_y, geno_ind * sizeof(double)));
  cutilSafeCall(hipMemcpy(d_y, y, geno_ind * sizeof(double), 
			   hipMemcpyHostToDevice));

  hipEventCreate(&start);
  hipEventCreate(&stopKernel);
  hipEventCreate(&stopMax);
  return 0;
}

//! @todo convert for gpu
void copyUpdateToDevice(unsigned id, unsigned iteration,  
			unsigned geno_count, unsigned n,
			char *d_snpMask, 
			int maxFIndex, double *d_Xtsnp, 
			size_t d_XtsnpPitch,
			const FortranMatrix &XtXi,
			const vector<double> &Xty){
  
  if(maxFIndex >= 0){
#ifdef _DEBUG
      cout << "iteration " << iteration << " id " << id 
	   << " masking index " << maxFIndex << " on device" 
	   << endl;
#endif
      cutilSafeCall(hipMemsetAsync(d_snpMask + maxFIndex, 1, 1));
#ifdef _DEBUG
    unsigned maskVal;
    cutilSafeCall(hipMemcpy(&maskVal, d_snpMask + maxFIndex,
			     sizeof(char), hipMemcpyDeviceToHost));
      cout << "iteration " << iteration << " id " << id 
	   << " mask index " << maxFIndex << ": "
	   << maskVal << endl;
#endif
  }
  
  //! copy updated XtSNP to GPU
  /*! @todo fix for GPU
  cutilSafeCall(hipMemcpy2D(d_Xtsnp + n, 
			     d_XtsnpPitch,
			     &XtSNP(n, (unsigned)0), 
			     (n + 1) * sizeof(double),
			     sizeof(double),
			     geno_count,
			     hipMemcpyHostToDevice));
  */

  // update GPU G (const mem)
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_G), &XtXi.values[0], 
				   (n + 1) * (n + 1) * sizeof(double)));
  
  // update GPU Xty (const mem)
  // call fails unless we update the whole thing
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Xty), &Xty[0], (n + 1) * sizeof(double)));
}

float getGPUCompTime(){
  float computation_elapsed_time;
  hipEventElapsedTime(&computation_elapsed_time, start, stopKernel);
  return computation_elapsed_time / 1000.0f;
}

float getGPUMaxTime(){
  float computation_elapsed_time;
  hipEventElapsedTime(&computation_elapsed_time, stopKernel, stopMax);
  return computation_elapsed_time / 1000.0f;
}

void getMaxFGPU(unsigned id, unsigned iteration, unsigned geno_count, 
	     vector<float> &Fval, 
	     unsigned maxFIndex, float *d_f){
#ifndef _DEBUG
    cutilSafeCall(hipMemcpy(&Fval[maxFIndex], &d_f[maxFIndex], sizeof(float),
			     hipMemcpyDeviceToHost));
#endif
}
